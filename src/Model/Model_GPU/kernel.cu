
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n_particles){
		for(int j = 0; j < n_particles; j++){
			if(i != j)
			{
				const float diffx = positionsGPU[j].x - positionsGPU[i].x;
				const float diffy = positionsGPU[j].y - positionsGPU[i].y;
				const float diffz = positionsGPU[j].z - positionsGPU[i].z;
				
				float dij = diffx * diffx + diffy * diffy + diffz * diffz;

				if (dij < EPS)
				{
					dij = 10.0;
				}
				else
				{
					dij = rsqrtf(dij);
					dij = 10.0 * (dij * dij * dij);
				}
				accelerationsGPU[i].x += diffx * dij * massesGPU[j];
				accelerationsGPU[i].y += diffy * dij * massesGPU[j];
				accelerationsGPU[i].z += diffz * dij * massesGPU[j];
			}
		}
	}
	
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	velocitiesGPU[i].x += accelerationsGPU[i].x * 2.0f;
	velocitiesGPU[i].y += accelerationsGPU[i].y * 2.0f;
	velocitiesGPU[i].z += accelerationsGPU[i].z * 2.0f;

	positionsGPU[i].x += velocitiesGPU[i].x * DIFF_T ;
	positionsGPU[i].y += velocitiesGPU[i].y * DIFF_T ;
	positionsGPU[i].z += velocitiesGPU[i].z * DIFF_T ;
}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU